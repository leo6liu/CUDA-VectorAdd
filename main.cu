//   _____
//  /     \             file: main.cu
//  vvvvvvv  /|__/|   author: Leo Battalora
//     I   /O,O   |
//     I /_____   |      /|/|
//    J|/^ ^ ^ \  |    /00  |    _//|
//     |^ ^ ^ ^ |W|   |/^^\ |   /oo |
//      \m___m__|_|    \m_m_|   \mm_|
//
// An example program which detects number of CUDA compute-capable devices,
// generates two float vectors of length 2^28 with values[0,1], adds them using
// the available CUDA compute-capable devices, and compares the result with a
// CPU solution by calculating the total error. 
//

// include files
//
#include "header.cuh"

// ----------------------------------------------------------------------------
// function: main
//
int
main (int argc, char **argv)
{
  // get number of CUDA compute-capable devices
  //
  int n_gpus;
  hipGetDeviceCount(&n_gpus);
  printf("INFO: %d CUDA compute-capable devices detected.\n", n_gpus);
  
  // exit program if no CUDA compute-capable devices found
  //
  if (n_gpus == 0) {
    printf("STATUS: Exiting progam... (no CUDA-enabled devices found)\n");
    return (0);
  }

  // print length and type of vectors
  //
  printf("INFO: This program adds two float vectors (A + B = C) of length %d.\n", 
	 VECTOR_LENGTH);

  // declare and allocate host vectors for a + b = c
  //
  float *a = (float *)malloc(VECTOR_LENGTH * sizeof(float));
  float *b = (float *)malloc(VECTOR_LENGTH * sizeof(float));
  float *c_cpu = (float *)malloc(VECTOR_LENGTH * sizeof(float));
  float *c_gpu = (float *)malloc(VECTOR_LENGTH * sizeof(float));

  // use current time as seed for random generator
  //
  srand(time(NULL));

  // initialize values for a and b
  //
  printf("STATUS: Initializing values for vector A...\n");
  vec_rand_init(a, VECTOR_LENGTH);
  printf("STATUS: Initializing values for vector B...\n");
  vec_rand_init(b, VECTOR_LENGTH);

  // print first five values of a and b
  //
  printf("DEBUG: Vector A:\n");
  vec_print(stdout, a, 3);
  printf("   ...\n");
  printf("DEBUG: Vector B:\n");
  vec_print(stdout, b, 3);
  printf("   ...\n");
  
  // calculate entrywise sum on GPU(s)
  //
  printf("STATUS: Calculating entrywise sum on GPU...\n");
  vec_add_gpu(a, b, c_gpu, VECTOR_LENGTH, n_gpus);

  // calcuate entrywise sum on CPU
  //
  printf("STATUS: Calculating entrywise sum on CPU...\n");
  vec_add_cpu(a, b, c_cpu, VECTOR_LENGTH);
  
  // print first five values of c_gpu
  //
  printf("DEBUG: Vector C (GPU):\n");
  vec_print(stdout, c_gpu, 3);
  printf("   ...\n");

  // print first five values of c_cpu
  //
  printf("DEBUG: Vector C (CPU):\n");
  vec_print(stdout, c_cpu, 3);
  printf("   ...\n");
  
  // calculate error between CPU and GPU results
  //
  printf("STATUS: Calculating total error between CPU and GPU results...\n");
  float error = vec_error(c_cpu, c_gpu, VECTOR_LENGTH);
  printf("INFO: Total error: %f\n", error);

  // deallocate host vectors
  //
  free(a);
  free(b);
  free(c_cpu);
  free(c_gpu);
  
  // exit normally
  //
  return (0);
}
