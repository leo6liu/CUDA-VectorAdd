//   _____
//  /     \             file: main.cu
//  vvvvvvv  /|__/|   author: Leo Battalora
//     I   /O,O   |
//     I /_____   |      /|/|
//    J|/^ ^ ^ \  |    /00  |    _//|
//     |^ ^ ^ ^ |W|   |/^^\ |   /oo |
//      \m___m__|_|    \m_m_|   \mm_|
//
// An example program which detects number of CUDA compute-capable devices,
// generates two float vectors of length 2^28 with values[0,1], adds them using
// the available CUDA compute-capable devices, and compares the result with a
// CPU solution by calculating the total error. 
//

// include files
//
#include "header.cuh"

//=============================================================================
//
// setup argument parser
//
//=============================================================================

// define argp global varaibles
//
const char *argp_program_version = "CUDA VectorAdd 1.0";
const char *argp_program_bug_address = "<leo6@temple.edu>";

// define doc (arg 4 of argp)
//
static char doc[] = "CUDA VectorAdd -- a program which scales across multiple \
GPUs to perform vector addition";

// define options (arg 1 of argp)
//   fields: {NAME, KEY, ARG, FLAGS, DOC}
//
struct argp_option options[] =
  {
    { "blocks-per-gpu", 'b', "LEN", 0,
      "Specify the number of blocks per GPU (defaults to either the the number of elements each GPU needs to process divided by the threads per block or the GPU's maxGridSize[0])" },
    { "vector-length", 'N', "LEN", 0, "Specify the vector lengths" },
    { "threads-per-block", 't', "LEN", 0,
      "Specify the number of threads per GPU block (defaults to maxThreadsPerBlock of GPU)" },
    { "verbose", 'v', 0, 0, "Explains what is being done" },
    { 0 }
  };

// declare arguments structure (used by main to communicate with parse_opt)
//
struct arguments
{
  int blocks;
  int N;
  int threads;
  int verbose;
};

// define parse_opt function (arg 2 of argp)
//
static error_t
parse_opt (int key, char *arg, struct argp_state *state)
{
  struct arguments *arguments = (struct arguments *)state->input;
  switch (key) {
  case 'b': // --blocks-per-gpu=LEN
    arguments->blocks = atoi(arg);
    break;
  case 'N': // --vector-length=LEN
    arguments->N = atoi(arg);
    break;
  case 't': // --threads-per-block=LEN
    arguments->threads = atoi(arg);
    break;
  case 'v': // --verbose
    arguments->verbose = 1;
    break;
  default:
    return ARGP_ERR_UNKNOWN;
  }
  return (0);
}

// define argp structure (arg 1 of argp_parse function in main)
//
static struct argp argp = { options, parse_opt, 0, doc };

// ----------------------------------------------------------------------------
// function: main
//
int
main (int argc, char **argv)
{
  // set default argument values
  //
  struct arguments args;
  args.blocks = -1; // determine blocks based on GPU properties
  args.N = VECTOR_LENGTH;
  args.threads = -1; // determine threads based on GPU properties
  args.verbose = 0;

  // parse for arguments and options
  //
  argp_parse (&argp, argc, argv, 0, 0, &args);
  
  // get number of CUDA compute-capable devices
  //
  int n_gpus;
  hipGetDeviceCount(&n_gpus);
  printf("INFO: %d CUDA compute-capable devices detected.\n", n_gpus);
  
  // exit program if no CUDA compute-capable devices found
  //
  if (n_gpus == 0) {
    printf("STATUS: Exiting progam... (no CUDA-enabled devices found)\n");
    return (0);
  }

  // print length and type of vectors
  //
  if (args.verbose == 1) {
    printf("INFO: This program adds two float vectors (A + B = C) of length %d.\n",
	   args.N);
  }
  
  // declare and allocate host vectors for a + b = c
  //
  float *a = (float *)malloc(args.N * sizeof(float));
  float *b = (float *)malloc(args.N * sizeof(float));
  float *c_cpu = (float *)malloc(args.N * sizeof(float));
  float *c_gpu = (float *)malloc(args.N * sizeof(float));
  
  // use current time as seed for random generator
  //
  srand(time(NULL));
  
  // initialize values for a and b
  //
  if (args.verbose == 1) {
    printf("STATUS: Initializing values for vector A...\n");
  }
  vec_rand_init(a, args.N);
  if (args.verbose == 1) {
    printf("STATUS: Initializing values for vector B...\n");
  }
  vec_rand_init(b, args.N);
  
  // print first five values of a and b
  //
  if (args.verbose == 1) {
    printf("DEBUG: Vector A:\n");
    vec_print(stdout, a, 3);
    printf("   ...\n");
    printf("DEBUG: Vector B:\n");
    vec_print(stdout, b, 3);
    printf("   ...\n");
  }
  
  // calculate entrywise sum on GPU(s)
  //
  if (args.verbose == 1) {
    printf("STATUS: Calculating entrywise sum on GPU...\n");
  }
  vec_add_gpu(a, b, c_gpu, args.N, n_gpus, args.threads, args.blocks);
  
  // calcuate entrywise sum on CPU
  //
  if (args.verbose == 1) {
    printf("STATUS: Calculating entrywise sum on CPU...\n");
  }
  vec_add_cpu(a, b, c_cpu, args.N);
  
  // print first five values of c_gpu
  //
  if (args.verbose == 1) {
    printf("DEBUG: Vector C (GPU):\n");
    vec_print(stdout, c_gpu, 3);
    printf("   ...\n");
  }

  // print first five values of c_cpu
  //
  if (args.verbose == 1) {
    printf("DEBUG: Vector C (CPU):\n");
    vec_print(stdout, c_cpu, 3);
    printf("   ...\n");
  }
  
  // calculate error between CPU and GPU results
  //
  if (args.verbose == 1) {
    printf("STATUS: Calculating total error between CPU and GPU results...\n");
  }
  float error = vec_error(c_cpu, c_gpu, args.N);
  printf("INFO: Total error: %f\n", error);

  // deallocate host vectors
  //
  free(a);
  free(b);
  free(c_cpu);
  free(c_gpu);
  
  // exit normally
  //
  return (0);
}
